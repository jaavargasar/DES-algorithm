
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>

using namespace std;

typedef unsigned long long ull;

const ull MAX = 10;


ull LnRnBlocks[17*2]; // from l0r0 to l16r16

ull CnDnBlocks[17*2]; //from c0d0 to c16d16

ull keysBlocks[16];  //from key[1] = k0 to key[16] = k15

ull allCipherDES[1000000];

ull Rotations[16] = {
    1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1
};

int PC1[56] = {
   57, 49, 41, 33, 25, 17,  9,
    1, 58, 50, 42, 34, 26, 18,
   10,  2, 59, 51, 43, 35, 27,
   19, 11,  3, 60, 52, 44, 36,
   63, 55, 47, 39, 31, 23, 15,
    7, 62, 54, 46, 38, 30, 22,
   14,  6, 61, 53, 45, 37, 29,
   21, 13,  5, 28, 20, 12,  4
};

int PC2[48] = {
   14, 17, 11, 24,  1,  5,
    3, 28, 15,  6, 21, 10,
   23, 19, 12,  4, 26,  8,
   16,  7, 27, 20, 13,  2,
   41, 52, 31, 37, 47, 55,
   30, 40, 51, 45, 33, 48,
   44, 49, 39, 56, 34, 53,
   46, 42, 50, 36, 29, 32
};

int IniPer[64] = {
    58, 50, 42, 34, 26, 18, 10,  2,
    60, 52, 44, 36, 28, 20, 12,  4,
    62, 54, 46, 38, 30, 22, 14,  6,
    64, 56, 48, 40, 32, 24, 16,  8,
    57, 49, 41, 33, 25, 17,  9,  1,
    59, 51, 43, 35, 27, 19, 11,  3,
    61, 53, 45, 37, 29, 21, 13,  5,
    63, 55, 47, 39, 31, 23, 15,  7
};

int reverseIniPer[64] ={
    40, 8, 48, 16, 56, 24, 64, 32,
    39, 7, 47, 15, 55, 23, 63, 31,
    38, 6, 46, 14, 54, 22, 62, 30,
    37, 5, 45, 13, 53, 21, 61, 29,
    36, 4, 44, 12, 52, 20, 60, 28,
    35, 3, 43, 11, 51, 19, 59, 27,
    34, 2, 42, 10, 50, 18, 58, 26,
    33, 1, 41,  9, 49, 17, 57, 25
};

int Expansion[48] ={
    32, 1,  2,   3, 4,  5,
    4,  5,  6,   7, 8,  9,
    8,  9,  10, 11, 12, 13,
    12, 13, 14, 15, 16, 17,
    16, 17, 18, 19, 20, 21,
    20, 21, 22, 23, 24, 25,
    24, 25, 26, 27, 28, 29,
    28, 29, 30, 31, 32,  1
};

int Pbox[32] ={
    16,  7, 20, 21,
    29, 12, 28, 17,
     1, 15, 23, 26,
     5, 18, 31, 10,
     2,  8, 24, 14,
    32, 27,  3,  9,
    19, 13, 30,  6,
    22, 11,  4, 25
};

int Sbox[512] = { //8*4*16
    
    14,  4, 13,  1,  2, 15, 11,  8,  3, 10,  6, 12,  5,  9,  0,  7,
     0, 15,  7,  4, 14,  2, 13,  1, 10,  6, 12, 11,  9,  5,  3,  8,
     4,  1, 14,  8, 13,  6,  2, 11, 15, 12,  9,  7,  3, 10,  5,  0,
    15, 12,  8,  2,  4,  9,  1,  7,  5, 11,  3, 14, 10,  0,  6, 13,
    15,  1,  8, 14,  6, 11,  3,  4,  9,  7,  2, 13, 12,  0,  5, 10,
     3, 13,  4,  7, 15,  2,  8, 14, 12,  0,  1, 10,  6,  9, 11,  5,
     0, 14,  7, 11, 10,  4, 13,  1,  5,  8, 12,  6,  9,  3,  2, 15,
    13,  8, 10,  1,  3, 15,  4,  2, 11,  6,  7, 12,  0,  5, 14,  9,
    10,  0,  9, 14,  6,  3, 15,  5,  1, 13, 12,  7, 11,  4,  2,  8,
    13,  7,  0,  9,  3,  4,  6, 10,  2,  8,  5, 14, 12, 11, 15,  1,
    13,  6,  4,  9,  8, 15,  3,  0, 11,  1,  2, 12,  5, 10, 14,  7,
     1, 10, 13,  0,  6,  9,  8,  7,  4, 15, 14,  3, 11,  5,  2, 12,
     7, 13, 14,  3,  0,  6,  9, 10,  1,  2,  8,  5, 11, 12,  4, 15,
    13,  8, 11,  5,  6, 15,  0,  3,  4,  7,  2, 12,  1, 10, 14,  9,
    10,  6,  9,  0, 12, 11,  7, 13, 15,  1,  3, 14,  5,  2,  8,  4,
     3, 15,  0,  6, 10,  1, 13,  8,  9,  4,  5, 11, 12,  7,  2, 14,
     2, 12,  4,  1,  7, 10, 11,  6,  8,  5,  3, 15, 13,  0, 14,  9,
    14, 11,  2, 12,  4,  7, 13,  1,  5,  0, 15, 10,  3,  9,  8,  6,
     4,  2,  1, 11, 10, 13,  7,  8, 15,  9, 12,  5,  6,  3,  0, 14,
    11,  8, 12,  7,  1, 14,  2, 13,  6, 15,  0,  9, 10,  4,  5,  3,
    12,  1, 10, 15,  9,  2,  6,  8,  0, 13,  3,  4, 14,  7,  5, 11,
    10, 15,  4,  2,  7, 12,  9,  5,  6,  1, 13, 14,  0, 11,  3,  8,
     9, 14, 15,  5,  2,  8, 12,  3,  7,  0,  4, 10,  1, 13, 11,  6,
     4,  3,  2, 12,  9,  5, 15, 10, 11, 14,  1,  7,  6,  0,  8, 13,
     4, 11,  2, 14, 15,  0,  8, 13,  3, 12,  9,  7,  5, 10,  6,  1,
    13,  0, 11,  7,  4,  9,  1, 10, 14,  3,  5, 12,  2, 15,  8,  6,
     1,  4, 11, 13, 12,  3,  7, 14, 10, 15,  6,  8,  0,  5,  9,  2,
     6, 11, 13,  8,  1,  4, 10,  7,  9,  5,  0, 15, 14,  2,  3, 12,
    13,  2,  8,  4,  6, 15, 11,  1, 10,  9,  3, 14,  5,  0, 12,  7,
     1, 15, 13,  8, 10,  3,  7,  4, 12,  5,  6, 11,  0, 14,  9,  2,
     7, 11,  4,  1,  9, 12, 14,  2,  0,  6, 10, 13, 15,  3,  5,  8,
     2,  1, 14,  7,  4, 10,  8, 13, 15, 12,  9,  0,  3,  5,  6, 11
 };

ull iniKey[8] = {
    0x13,0x34,0x57,0x79,0x9B,0xBC,0xDF,0xF1};

ull message[8] = {    
    0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF};



__host__ __device__ ull generateKeyPlus(ull *d_iniKey, int *d_PC1){
    ull keyPlus=0L;
    for(int i=56-1;i>=0;i--){
        if( d_iniKey[ d_PC1[i]/8 ] & (1 << ( ( 64-d_PC1[i]) % 8 ) ) ){
            keyPlus|=( 1LL<< (55-i)*1L );
        }
    }
    return keyPlus;
}

__host__ __device__ ull* splitKeyPlus(ull keyPlus){
    ull c0=0L, d0=0L;

    for(int i=0;i<28;i++){
        if(keyPlus & (1LL<<i*1L ) ) d0|=(1LL<<i*1L);
        if(keyPlus & (1LL<< i*1L +28L ) ) c0|=(1LL<<i*1L);
    }
    ull new_array[2]= {c0,d0};
    return new_array;
}

__host__ __device__ ull* splitIniPer(ull codeIniPer){
    ull l0=0L, r0=0L;

    for(int i=0;i<32;i++){
        if(codeIniPer & (1LL<<i*1L ) ) r0|=(1LL<<i*1L);
        if(codeIniPer & (1LL<< i*1L +32L ) ) l0|=(1LL<<i*1L);
    }
    ull new_array[2] ={ l0,r0};
    return new_array;
}

__host__ __device__ void generateCnDnBlocks( ull *seedKey,ull *d_CnDnBlocks, ull *d_Rotations){
    d_CnDnBlocks[0] = seedKey[0];
    d_CnDnBlocks[1] = seedKey[1];
    ull cn ,dn, newCn, newDn;
    ull getOnCn, getOnDn;

    for(int i=1;i<=16;i++){
        
        getOnCn = 0L;
        getOnDn = 0L;

        cn = d_CnDnBlocks[(i*2+0)-2];
        dn = d_CnDnBlocks[(i*2+1)-2];

        for(ull j=0;j< d_Rotations[ i-1 ];j++){

            if(  cn & (1 << (27-j) )  ) getOnCn|= 1LL << (d_Rotations[ i-1 ]==1 ? j: 1-j) ;
            if(  dn & (1 << (27-j) )  ) getOnDn|= 1LL << (d_Rotations[ i-1 ]==1 ? j: 1-j);

        }

        newCn = cn << d_Rotations[ i-1 ];
        newDn = dn << d_Rotations[ i-1 ];

        for(ull j=0; j< d_Rotations[ i-1 ] ;j++){
            newCn &= ~(1<< (28+j) );
            newDn &= ~(1<< (28+j) );
        }
        newCn |= ( getOnCn );
        newDn |= ( getOnDn );
        d_CnDnBlocks[ (i*2)+0 ] = newCn;
        d_CnDnBlocks[ (i*2)+1 ] = newDn;
    }

}

__host__ __device__ ull joinCnDn(ull cn, ull dn){ return (cn<<28) | dn; }

__host__ __device__ void generateKeysBlocks(ull *d_CnDnBlocks, int *d_PC2,ull *d_keysBlocks){
    ull cnDn, keyn;

    for(int i=1;i<=16;i++){
        cnDn = joinCnDn( d_CnDnBlocks[i*2+0], d_CnDnBlocks[i*2+1] );
        keyn = 0L;

        for(int j=48-1;j>=0;j--){
            if( cnDn & ( 1LL << (56-d_PC2[j])*1L ) ) {
                 keyn|= ( 1LL<< (47-j)*1L );
            }
        }
        d_keysBlocks[i-1] = keyn;
    }

}

__host__ __device__ ull generateIniPer(int *d_IniPer, ull *d_message){
    ull keyPlus=0L;

    for(int i=64-1;i>=0;i--){
        if( d_message[ (d_IniPer[i]/8) >=8 ? 7: (d_IniPer[i]/8) ]  & (1LL << ( ( 64-d_IniPer[i]) % 8 ) ) ){
            keyPlus|=( 1LL<< (63-i)*1L );
        }
    }
    
    return keyPlus;
}

__host__ __device__ ull expandRn(ull Rn,int *d_Expansion){
    //from a Rn 32 bit to a Kn 48 bit
    ull exRn=0L;
    for(int j=48-1;j>=0;j--){
            if( Rn & ( 1LL << (32-d_Expansion[j])*1L ) ) {
                 exRn|= ( 1LL<< (47-j)*1L );
            }
    }
    return exRn;
}

__host__ __device__ ull xorOperation(ull En, ull Kn){
    return (Kn ^ En);
}

__host__ __device__ ull getSboxNumber(int Bn, int k, int *d_Sbox){

    int row=0,col=0;
    if( Bn & 1<<0 ) row |= ( 1<<0);
    if( Bn & 1<<6-1) row |=( 1<<1 );

    for(int i=1;i<=4;i++){
        if( Bn & 1<<i ) col |=(1<<(i-1));
    }

    return d_Sbox[ (k*4*16)+(row*16)+col ];
}


__host__ __device__ ull generateSboxCombination(ull Bn,int *d_Sbox){

    int Bbox[8];
    ull sbBox[8];
    int number=0, k=7;
    ull snBnChain=0L;
    ull step=28L;

    for(int i=0;i<=48;i++ ){

        if( i%6==0 && i>=6){
            

            Bbox[ k-- ] = number;
            number = 0;
        }
        if( Bn & (1LL<<i*1L) ){
            number |= (1LL<< ( (i%6)*1L ) );
        }
    }

    for(int i=0;i<8;i++){
        sbBox[i] = getSboxNumber( Bbox[i], i,d_Sbox);

       
    }

    
    for(int i=0;i<8;i++){
        snBnChain |= (sbBox[i]<< step);
        step-=4;
    }

    return snBnChain;

}

__host__ __device__ ull generateFalgorithm(ull snBn, int *d_Pbox){

    ull fn=0L;
    for(int j=32-1;j>=0;j--){
            if( snBn & ( 1LL << (32-d_Pbox[j])*1L ) ) {
                 fn|= ( 1LL<< (31-j)*1L );
            }
    }
    return fn;
}

__host__ __device__ void generateLnRnBlocks(ull *L0R0,ull *d_LnRnBlocks, ull *d_keysBlocks,int *d_Expansion, int *d_Sbox,int *d_Pbox){

    d_LnRnBlocks[0] = L0R0[0];
    d_LnRnBlocks[1] = L0R0[1];
    ull fn;

    for(int time=1; time<=16;time++){

        ull Ln = d_LnRnBlocks[ (time*2+0)-2 ];
        ull Rn = d_LnRnBlocks[ (time*2+1)-2 ];
       
        ull snBn = 
            generateSboxCombination( xorOperation( expandRn( Rn, d_Expansion ),d_keysBlocks[ time-1 ] ),d_Sbox );
        
        fn = generateFalgorithm(snBn,d_Pbox);

        d_LnRnBlocks[ (time*2+0) ] = Rn;
        d_LnRnBlocks[ (time*2+1) ] = (Ln ^ fn);

    }

}

__host__ __device__ ull reverseLnRn( ull *LnRn, ull *LnRn1){
    ull Ln = *LnRn;
    ull Rn = *LnRn1;

    return ( Rn<<32L) | Ln;
}

__host__ __device__ ull generateCipherMessage( ull RnLn, int *d_reverseIniPer ){

    ull cipher=0L;
    for(int j=64-1;j>=0;j--){
            if( RnLn & ( 1LL << (64-d_reverseIniPer[j])*1L ) ) {
                 cipher|= ( 1LL<< (63-j)*1L );
            }
    }
    return cipher;
}


__global__ void cipherDES(
    ull *d_LnRnBlocks,
    ull *d_CnDnBlocks,
    ull *d_keysBlocks,
    ull *d_allCipherDES,
    ull *d_Rotations,
    int *d_PC1,
    int *d_PC2,
    int *d_IniPer,
    int *d_reverseIniPer,
    int *d_Expansion,
    int *d_Pbox,
    int *d_Sbox,
    ull *d_iniKey,
    ull *d_message
){
    ull *keyHalves = splitKeyPlus( generateKeyPlus(d_iniKey,d_PC1) );
    generateCnDnBlocks( keyHalves,d_CnDnBlocks,d_Rotations );
    generateKeysBlocks(d_CnDnBlocks,d_PC2,d_keysBlocks);
    ull *iniPerHalves = splitIniPer(generateIniPer(d_IniPer,d_message) ); //got L0 and R0
    

    generateLnRnBlocks( iniPerHalves,d_LnRnBlocks, d_keysBlocks, d_Expansion, d_Sbox, d_Pbox);

    ull revLnRn = reverseLnRn( &d_LnRnBlocks[16*2+0],&d_LnRnBlocks[16*2+1] );
   
    ull cipherMessage = generateCipherMessage( revLnRn,d_reverseIniPer );
    // // printf("cipher: %llu\n",cipherMessage);
    // // fflush(stdout);

    printf("Hex Cipher: %llX\n", cipherMessage);
   
    // return cipherMessage;
}

int main(){

    //host and devices copies
    ull *d_LnRnBlocks;//17 size
    ull *d_CnDnBlocks;//17 size
    ull *d_keysBlocks;//16 size
    ull *d_allCipherDES;//10^6 size
    ull *d_Rotations;//16 size
    int *d_PC1;//56 size
    int *d_PC2;//48 size
    int *d_IniPer;//64 size
    int *d_reverseIniPer;//64 size
    int *d_Expansion;//48 size
    int *d_Pbox;//32 size
    int *d_Sbox;//8*8*16 size [8][4][16]
    ull *d_iniKey;//8 size
    ull *d_message;//8 size

    //size of host and device copies
    int sd_LnRnBlocks = 17 * 2 * sizeof(ull);
    int sd_CnDnBlocks = 17 * 2 * sizeof(ull);
    int sd_keysBlocks = 16 * sizeof(ull);
    int sd_allCipherDES = 1000000 * sizeof(ull);
    int sd_Rotations = 16 * sizeof(ull);
    int sd_PC1 = 56 * sizeof(int);
    int sd_PC2 = 48 * sizeof(int);
    int sd_IniPer = 64 * sizeof(int);
    int sd_reverseIniPer = 64 * sizeof(int);
    int sd_Expansion = 48 * sizeof(int);
    int sd_Pbox = 32 * sizeof(int);
    int sd_Sbox = 512 * sizeof(int);
    int sd_iniKey = 8 * sizeof(ull);
    int sd_message = 8 * sizeof(ull);

    //alloc space for host and device copies
    hipMalloc( (void **)&d_LnRnBlocks, sd_LnRnBlocks );
    hipMalloc( (void **)&d_CnDnBlocks, sd_CnDnBlocks );
    hipMalloc( (void **)&d_keysBlocks, sd_keysBlocks );
    hipMalloc( (void **)&d_allCipherDES, sd_allCipherDES );
    hipMalloc( (void **)&d_Rotations, sd_Rotations );
    hipMalloc( (void **)&d_PC1, sd_PC1 );
    hipMalloc( (void **)&d_PC2, sd_PC2 );
    hipMalloc( (void **)&d_IniPer, sd_IniPer );
    hipMalloc( (void **)&d_reverseIniPer, sd_reverseIniPer );
    hipMalloc( (void **)&d_Expansion, sd_Expansion );
    hipMalloc( (void **)&d_Pbox, sd_Pbox );
    hipMalloc( (void **)&d_Sbox, sd_Sbox );
    hipMalloc( (void **)&d_iniKey, sd_iniKey );
    hipMalloc( (void **)&d_message, sd_message );

    //copy inputs to device
    hipMemcpy(d_LnRnBlocks,LnRnBlocks,sd_LnRnBlocks,hipMemcpyHostToDevice);
    hipMemcpy(d_CnDnBlocks,CnDnBlocks,sd_CnDnBlocks,hipMemcpyHostToDevice);
    hipMemcpy(d_keysBlocks,keysBlocks,sd_keysBlocks,hipMemcpyHostToDevice);
    hipMemcpy(d_allCipherDES,allCipherDES,sd_allCipherDES,hipMemcpyHostToDevice);
    hipMemcpy(d_Rotations,Rotations,sd_Rotations,hipMemcpyHostToDevice);
    hipMemcpy(d_PC1,PC1,sd_PC1,hipMemcpyHostToDevice);
    hipMemcpy(d_PC2,PC2,sd_PC2,hipMemcpyHostToDevice);
    hipMemcpy(d_IniPer,IniPer,sd_IniPer,hipMemcpyHostToDevice);
    hipMemcpy(d_reverseIniPer,reverseIniPer,sd_reverseIniPer,hipMemcpyHostToDevice);
    hipMemcpy(d_Expansion,Expansion,sd_Expansion,hipMemcpyHostToDevice);
    hipMemcpy(d_Pbox,Pbox,sd_Pbox,hipMemcpyHostToDevice);
    hipMemcpy(d_Sbox,Sbox,sd_Sbox,hipMemcpyHostToDevice);
    hipMemcpy(d_iniKey,iniKey,sd_iniKey,hipMemcpyHostToDevice);
    hipMemcpy(d_message,message,sd_message,hipMemcpyHostToDevice);
    
    //launch kernel     ----------- HERES THE MAGIC ---------
    cipherDES<<<1,1>>>(
        d_LnRnBlocks,
        d_CnDnBlocks,
        d_keysBlocks,
        d_allCipherDES,
        d_Rotations,
        d_PC1,
        d_PC2,
        d_IniPer,
        d_reverseIniPer,
        d_Expansion,
        d_Pbox,
        d_Sbox,
        d_iniKey,
        d_message
    );

    //free cuda space
    hipFree(d_LnRnBlocks);
    hipFree(d_CnDnBlocks);
    hipFree(d_keysBlocks);
    hipFree(d_allCipherDES);
    hipFree(d_Rotations);
    hipFree(d_PC1);
    hipFree(d_PC2);
    hipFree(d_IniPer);
    hipFree(d_reverseIniPer);
    hipFree(d_Expansion);
    hipFree(d_Pbox);
    hipFree(d_Sbox);
    hipFree(d_iniKey);
    hipFree(d_message);

    return 0;
}


// L16 01000011010000100011001000110100    1128411700
// R16 00001010010011001101100110010101    172808597

// revL16R16 0000101001001100110110011001010101000011010000100011001000110100  742207273711055412

// cipher 1000010111101000000100110101010000001111000010101011010000000101     9648983453391827973