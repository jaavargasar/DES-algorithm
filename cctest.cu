
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>

using namespace std;

typedef unsigned long long ull;
typedef pair< ull , ull>  uull;

const ull MAX = 10;


uull LnRnBlocks[17]; // from l0r0 to l16r16

uull CnDnBlocks[17]; //from c0d0 to c16d16

ull keysBlocks[16];  //from key[1] = k0 to key[16] = k15

ull allCipherDES[1000000];

const ull Rotations[16] = {
    1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1
};

const int PC1[56] = {
   57, 49, 41, 33, 25, 17,  9,
    1, 58, 50, 42, 34, 26, 18,
   10,  2, 59, 51, 43, 35, 27,
   19, 11,  3, 60, 52, 44, 36,
   63, 55, 47, 39, 31, 23, 15,
    7, 62, 54, 46, 38, 30, 22,
   14,  6, 61, 53, 45, 37, 29,
   21, 13,  5, 28, 20, 12,  4
};

const int PC2[48] = {
   14, 17, 11, 24,  1,  5,
    3, 28, 15,  6, 21, 10,
   23, 19, 12,  4, 26,  8,
   16,  7, 27, 20, 13,  2,
   41, 52, 31, 37, 47, 55,
   30, 40, 51, 45, 33, 48,
   44, 49, 39, 56, 34, 53,
   46, 42, 50, 36, 29, 32
};

const int IniPer[64] = {
    58, 50, 42, 34, 26, 18, 10,  2,
    60, 52, 44, 36, 28, 20, 12,  4,
    62, 54, 46, 38, 30, 22, 14,  6,
    64, 56, 48, 40, 32, 24, 16,  8,
    57, 49, 41, 33, 25, 17,  9,  1,
    59, 51, 43, 35, 27, 19, 11,  3,
    61, 53, 45, 37, 29, 21, 13,  5,
    63, 55, 47, 39, 31, 23, 15,  7
};

const int reverseIniPer[64] ={
    40, 8, 48, 16, 56, 24, 64, 32,
    39, 7, 47, 15, 55, 23, 63, 31,
    38, 6, 46, 14, 54, 22, 62, 30,
    37, 5, 45, 13, 53, 21, 61, 29,
    36, 4, 44, 12, 52, 20, 60, 28,
    35, 3, 43, 11, 51, 19, 59, 27,
    34, 2, 42, 10, 50, 18, 58, 26,
    33, 1, 41,  9, 49, 17, 57, 25
};

const int Expansion[48] ={
    32, 1,  2,   3, 4,  5,
    4,  5,  6,   7, 8,  9,
    8,  9,  10, 11, 12, 13,
    12, 13, 14, 15, 16, 17,
    16, 17, 18, 19, 20, 21,
    20, 21, 22, 23, 24, 25,
    24, 25, 26, 27, 28, 29,
    28, 29, 30, 31, 32,  1
};

const int Pbox[32] ={
    16,  7, 20, 21,
    29, 12, 28, 17,
     1, 15, 23, 26,
     5, 18, 31, 10,
     2,  8, 24, 14,
    32, 27,  3,  9,
    19, 13, 30,  6,
    22, 11,  4, 25
};

const int Sbox[8][4][16] = {
   {
   {14,  4, 13,  1,  2, 15, 11,  8,  3, 10,  6, 12,  5,  9,  0,  7},
   { 0, 15,  7,  4, 14,  2, 13,  1, 10,  6, 12, 11,  9,  5,  3,  8},
   { 4,  1, 14,  8, 13,  6,  2, 11, 15, 12,  9,  7,  3, 10,  5,  0},
   {15, 12,  8,  2,  4,  9,  1,  7,  5, 11,  3, 14, 10,  0,  6, 13},
   },
 
   {
   {15,  1,  8, 14,  6, 11,  3,  4,  9,  7,  2, 13, 12,  0,  5, 10},
   { 3, 13,  4,  7, 15,  2,  8, 14, 12,  0,  1, 10,  6,  9, 11,  5},
   { 0, 14,  7, 11, 10,  4, 13,  1,  5,  8, 12,  6,  9,  3,  2, 15},
   {13,  8, 10,  1,  3, 15,  4,  2, 11,  6,  7, 12,  0,  5, 14,  9},
   },
 
   {
   {10,  0,  9, 14,  6,  3, 15,  5,  1, 13, 12,  7, 11,  4,  2,  8},
   {13,  7,  0,  9,  3,  4,  6, 10,  2,  8,  5, 14, 12, 11, 15,  1},
   {13,  6,  4,  9,  8, 15,  3,  0, 11,  1,  2, 12,  5, 10, 14,  7},
   { 1, 10, 13,  0,  6,  9,  8,  7,  4, 15, 14,  3, 11,  5,  2, 12},
   },
 
   {
   { 7, 13, 14,  3,  0,  6,  9, 10,  1,  2,  8,  5, 11, 12,  4, 15},
   {13,  8, 11,  5,  6, 15,  0,  3,  4,  7,  2, 12,  1, 10, 14,  9},
   {10,  6,  9,  0, 12, 11,  7, 13, 15,  1,  3, 14,  5,  2,  8,  4},
   { 3, 15,  0,  6, 10,  1, 13,  8,  9,  4,  5, 11, 12,  7,  2, 14},
   },
 
   {
   { 2, 12,  4,  1,  7, 10, 11,  6,  8,  5,  3, 15, 13,  0, 14,  9},
   {14, 11,  2, 12,  4,  7, 13,  1,  5,  0, 15, 10,  3,  9,  8,  6},
   { 4,  2,  1, 11, 10, 13,  7,  8, 15,  9, 12,  5,  6,  3,  0, 14},
   {11,  8, 12,  7,  1, 14,  2, 13,  6, 15,  0,  9, 10,  4,  5,  3},
   },
 
   {
   {12,  1, 10, 15,  9,  2,  6,  8,  0, 13,  3,  4, 14,  7,  5, 11},
   {10, 15,  4,  2,  7, 12,  9,  5,  6,  1, 13, 14,  0, 11,  3,  8},
   { 9, 14, 15,  5,  2,  8, 12,  3,  7,  0,  4, 10,  1, 13, 11,  6},
   { 4,  3,  2, 12,  9,  5, 15, 10, 11, 14,  1,  7,  6,  0,  8, 13},
   },
 
   {
   { 4, 11,  2, 14, 15,  0,  8, 13,  3, 12,  9,  7,  5, 10,  6,  1},
   {13,  0, 11,  7,  4,  9,  1, 10, 14,  3,  5, 12,  2, 15,  8,  6},
   { 1,  4, 11, 13, 12,  3,  7, 14, 10, 15,  6,  8,  0,  5,  9,  2},
   { 6, 11, 13,  8,  1,  4, 10,  7,  9,  5,  0, 15, 14,  2,  3, 12},
   },
   {
   {13,  2,  8,  4,  6, 15, 11,  1, 10,  9,  3, 14,  5,  0, 12,  7},
   { 1, 15, 13,  8, 10,  3,  7,  4, 12,  5,  6, 11,  0, 14,  9,  2},
   { 7, 11,  4,  1,  9, 12, 14,  2,  0,  6, 10, 13, 15,  3,  5,  8},
   { 2,  1, 14,  7,  4, 10,  8, 13, 15, 12,  9,  0,  3,  5,  6, 11},
   },
};

const ull iniKey[8] = {
    0x13,0x34,0x57,0x79,0x9B,0xBC,0xDF,0xF1};

const ull message[8] = {    
    0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF};



ull generateKeyPlus(){
    ull keyPlus=0L;
    for(int i=56-1;i>=0;i--){
        if( iniKey[ PC1[i]/8 ] & (1 << ( ( 64-PC1[i]) % 8 ) ) ){
            keyPlus|=( 1LL<< (55-i)*1L );
        }
    }
    return keyPlus;
}

uull splitKeyPlus(ull keyPlus){
    ull c0=0L, d0=0L;

    for(int i=0;i<28;i++){
        if(keyPlus & (1LL<<i*1L ) ) d0|=(1LL<<i*1L);
        if(keyPlus & (1LL<< i*1L +28L ) ) c0|=(1LL<<i*1L);
    }
    return make_pair( c0, d0);
}

uull splitIniPer(ull codeIniPer){
    ull l0=0L, r0=0L;

    for(int i=0;i<32;i++){
        if(codeIniPer & (1LL<<i*1L ) ) r0|=(1LL<<i*1L);
        if(codeIniPer & (1LL<< i*1L +32L ) ) l0|=(1LL<<i*1L);
    }
    return make_pair( l0, r0);
}

void generateCnDnBlocks( uull seedKey){
    CnDnBlocks[0] = seedKey;
    ull cn ,dn, newCn, newDn;
    ull getOnCn, getOnDn;

    for(int i=1;i<=16;i++){
        
        getOnCn = 0L;
        getOnDn = 0L;

        cn = CnDnBlocks[i-1].first ;
        dn = CnDnBlocks[i-1].second;

        for(ull j=0;j< Rotations[ i-1 ];j++){

            if(  cn & (1 << (27-j) )  ) getOnCn|= 1LL << (Rotations[ i-1 ]==1 ? j: 1-j) ;
            if(  dn & (1 << (27-j) )  ) getOnDn|= 1LL << (Rotations[ i-1 ]==1 ? j: 1-j);

        }

        newCn = cn << Rotations[ i-1 ];
        newDn = dn << Rotations[ i-1 ];

        for(ull j=0; j< Rotations[ i-1 ] ;j++){
            newCn &= ~(1<< (28+j) );
            newDn &= ~(1<< (28+j) );
        }
        newCn |= ( getOnCn );
        newDn |= ( getOnDn );
        CnDnBlocks[ i ] = make_pair( newCn, newDn);
    }

}

ull joinCnDn(ull cn, ull dn){ return (cn<<28) | dn; }

void generateKeysBlocks(){
    ull cnDn, keyn;

    for(int i=1;i<=16;i++){
        cnDn = joinCnDn( CnDnBlocks[i].first, CnDnBlocks[i].second );
        keyn = 0L;

        for(int j=48-1;j>=0;j--){
            if( cnDn & ( 1LL << (56-PC2[j])*1L ) ) {
                 keyn|= ( 1LL<< (47-j)*1L );
            }
        }
        keysBlocks[i-1] = keyn;
    }

}

ull generateIniPer(){
    ull keyPlus=0L;

    for(int i=64-1;i>=0;i--){
        if( message[ (IniPer[i]/8) >=8 ? 7: (IniPer[i]/8) ]  & (1LL << ( ( 64-IniPer[i]) % 8 ) ) ){
            keyPlus|=( 1LL<< (63-i)*1L );
        }
    }
    
    return keyPlus;
}

ull expandRn(ull Rn){
    //from a Rn 32 bit to a Kn 48 bit
    ull exRn=0L;
    for(int j=48-1;j>=0;j--){
            if( Rn & ( 1LL << (32-Expansion[j])*1L ) ) {
                 exRn|= ( 1LL<< (47-j)*1L );
            }
    }
    return exRn;
}

ull xorOperation(ull En, ull Kn){
    return (Kn ^ En);
}

ull getSboxNumber(int Bn, int k){

    int row=0,col=0;
    if( Bn & 1<<0 ) row |= ( 1<<0);
    if( Bn & 1<<6-1) row |=( 1<<1 );

    for(int i=1;i<=4;i++){
        if( Bn & 1<<i ) col |=(1<<(i-1));
    }

    return ( Sbox[k][row][col]*1LL);
}


ull generateSboxCombination(ull Bn){

    int Bbox[8];
    ull sbBox[8];
    int number=0, k=7;
    ull snBnChain=0L;
    ull step=28L;

    for(int i=0;i<=48;i++ ){

        if( i%6==0 && i>=6){
            

            Bbox[ k-- ] = number;
            number = 0;
        }
        if( Bn & (1LL<<i*1L) ){
            number |= (1LL<< ( (i%6)*1L ) );
        }
    }

    for(int i=0;i<8;i++){
        sbBox[i] = getSboxNumber( Bbox[i], i);

       
    }

    
    for(int i=0;i<8;i++){
        snBnChain |= (sbBox[i]<< step);
        step-=4;
    }

    return snBnChain;

}

ull generateFalgorithm(ull snBn){

    ull fn=0L;
    for(int j=32-1;j>=0;j--){
            if( snBn & ( 1LL << (32-Pbox[j])*1L ) ) {
                 fn|= ( 1LL<< (31-j)*1L );
            }
    }
    return fn;
}

void generateLnRnBlocks(uull L0R0){

    LnRnBlocks[0] = L0R0;
    ull fn;

    for(int time=1; time<=16;time++){

        ull Ln = LnRnBlocks[ time-1 ].first;
        ull Rn = LnRnBlocks[ time-1 ].second;
       
        ull snBn = 
            generateSboxCombination( xorOperation( expandRn( Rn ),keysBlocks[ time-1 ] ) );
        
        fn = generateFalgorithm(snBn);

        uull LnRn = make_pair( Rn, (Ln ^ fn) ); 
        LnRnBlocks[ time ] = LnRn;

    }

}

ull reverseLnRn( uull LnRn){
    ull Ln = LnRn.first;
    ull Rn = LnRn.second;

    return ( Rn<<32L) | Ln;
}

ull generateCipherMessage( ull RnLn ){

    ull cipher=0L;
    for(int j=64-1;j>=0;j--){
            if( RnLn & ( 1LL << (64-reverseIniPer[j])*1L ) ) {
                 cipher|= ( 1LL<< (63-j)*1L );
            }
    }
    return cipher;
}


__global__ void cipherDES(
    uull *LnRnBlocks,
    uull *CnDnBlocks,
    ull *keysBlocks,
    ull *allCipherDES,
    ull *Rotations,
    int *PC1,
    int *PC2,
    int *IniPer,
    int *reverseIniPer,
    int *Expansion,
    int *Pbox,
    int *Sbox,
    ull *iniKey,
    ull *message
){
    // uull keyHalves = splitKeyPlus( generateKeyPlus() );
    // generateCnDnBlocks( keyHalves );
    // generateKeysBlocks();
    // uull iniPerHalves = splitIniPer(generateIniPer() ); //got L0 and R0
    

    // generateLnRnBlocks( iniPerHalves );

    // ull revLnRn = reverseLnRn( LnRnBlocks[16] );
   
    // ull cipherMessage = generateCipherMessage( revLnRn );
    // // printf("cipher: %llu\n",cipherMessage);
    // // fflush(stdout);

    // // printf("Hex Cipher: %llX\n", cipherMessage);
    // // fflush(stdout);
    // return cipherMessage;
    printf("helloooooooooooooo there %i\n",PC1[10]);
}

int main(){

    //host and devices copies
    uull *d_LnRnBlocks;//17 size
    uull *d_CnDnBlocks;//17 size
    ull *d_keysBlocks;//16 size
    ull *d_allCipherDES;//10^6 size
    ull *d_Rotations;//16 size
    int *d_PC1;//56 size
    int *d_PC2;//48 size
    int *d_IniPer;//64 size
    int *d_reverseIniPer;//64 size
    int *d_Expansion;//48 size
    int *d_Pbox;//32 size
    int *d_Sbox;//8*8*16 size [8][4][16]
    ull *d_iniKey;//8 size
    ull *d_message;//8 size

    //size of host and device copies
    int sd_LnRnBlocks = 17 * sizeof(uull);
    int sd_CnDnBlocks = 17 * sizeof(uull);
    int sd_keysBlocks = 16 * sizeof(ull);
    int sd_allCipherDES = 1000000 * sizeof(ull);
    int sd_Rotations = 16 * sizeof(ull);
    int sd_PC1 = 56 * sizeof(int);
    int sd_PC2 = 48 * sizeof(int);
    int sd_IniPer = 64 * sizeof(int);
    int sd_reverseIniPer = 64 * sizeof(int);
    int sd_Expansion = 48 * sizeof(int);
    int sd_Pbox = 32 * sizeof(int);
    int sd_Sbox = 512 * sizeof(int);
    int sd_iniKey = 8 * sizeof(ull);
    int sd_message = 8 * sizeof(ull);

    //alloc space for host and device copies
    hipMalloc( (void **)&d_LnRnBlocks, sd_LnRnBlocks );
    hipMalloc( (void **)&d_CnDnBlocks, sd_CnDnBlocks );
    hipMalloc( (void **)&d_keysBlocks, sd_keysBlocks );
    hipMalloc( (void **)&d_allCipherDES, sd_allCipherDES );
    hipMalloc( (void **)&d_Rotations, sd_Rotations );
    hipMalloc( (void **)&d_PC1, sd_PC1 );
    hipMalloc( (void **)&d_PC2, sd_PC2 );
    hipMalloc( (void **)&d_IniPer, sd_IniPer );
    hipMalloc( (void **)&d_reverseIniPer, sd_reverseIniPer );
    hipMalloc( (void **)&d_Expansion, sd_Expansion );
    hipMalloc( (void **)&d_Pbox, sd_Pbox );
    hipMalloc( (void **)&d_Sbox, sd_Sbox );
    hipMalloc( (void **)&d_iniKey, sd_iniKey );
    hipMalloc( (void **)&d_message, sd_message );

    //copy inputs to device
    hipMemcpy(d_LnRnBlocks,LnRnBlocks,sd_LnRnBlocks,hipMemcpyHostToDevice);
    hipMemcpy(d_CnDnBlocks,CnDnBlocks,sd_CnDnBlocks,hipMemcpyHostToDevice);
    hipMemcpy(d_keysBlocks,keysBlocks,sd_keysBlocks,hipMemcpyHostToDevice);
    hipMemcpy(d_allCipherDES,allCipherDES,sd_allCipherDES,hipMemcpyHostToDevice);
    hipMemcpy(d_Rotations,Rotations,sd_Rotations,hipMemcpyHostToDevice);
    hipMemcpy(d_PC1,PC1,sd_PC1,hipMemcpyHostToDevice);
    hipMemcpy(d_PC2,PC2,sd_PC2,hipMemcpyHostToDevice);
    hipMemcpy(d_IniPer,IniPer,sd_IniPer,hipMemcpyHostToDevice);
    hipMemcpy(d_reverseIniPer,reverseIniPer,sd_reverseIniPer,hipMemcpyHostToDevice);
    hipMemcpy(d_Expansion,Expansion,sd_Expansion,hipMemcpyHostToDevice);
    hipMemcpy(d_Pbox,Pbox,sd_Pbox,hipMemcpyHostToDevice);
    hipMemcpy(d_Sbox,Sbox,sd_Sbox,hipMemcpyHostToDevice);
    hipMemcpy(d_iniKey,iniKey,sd_iniKey,hipMemcpyHostToDevice);
    hipMemcpy(d_message,message,sd_message,hipMemcpyHostToDevice);
    
    //launch kernel     ----------- HERES THE MAGIC ---------
    cipherDES<<<1,1>>>(
        d_LnRnBlocks,
        d_CnDnBlocks,
        d_keysBlocks,
        d_allCipherDES,
        d_Rotations,
        d_PC1,
        d_PC2,
        d_IniPer,
        d_reverseIniPer,
        d_Expansion,
        d_Pbox,
        d_Sbox,
        d_iniKey,
        d_message
    );

    //free cuda space
    hipFree(d_LnRnBlocks);
    hipFree(d_CnDnBlocks);
    hipFree(d_keysBlocks);
    hipFree(d_allCipherDES);
    hipFree(d_Rotations);
    hipFree(d_PC1);
    hipFree(d_PC2);
    hipFree(d_IniPer);
    hipFree(d_reverseIniPer);
    hipFree(d_Expansion);
    hipFree(d_Pbox);
    hipFree(d_Sbox);
    hipFree(d_iniKey);
    hipFree(d_message);

    return 0;
}


// L16 01000011010000100011001000110100    1128411700
// R16 00001010010011001101100110010101    172808597

// revL16R16 0000101001001100110110011001010101000011010000100011001000110100  742207273711055412

// cipher 1000010111101000000100110101010000001111000010101011010000000101     9648983453391827973