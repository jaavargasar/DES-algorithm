
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

const int PC1[56] = {
    57, 49, 41, 33, 25, 17,  9,
     1, 58, 50, 42, 34, 26, 18,
    10,  2, 59, 51, 43, 35, 27,
    19, 11,  3, 60, 52, 44, 36,
    63, 55, 47, 39, 31, 23, 15,
     7, 62, 54, 46, 38, 30, 22,
    14,  6, 61, 53, 45, 37, 29,
    21, 13,  5, 28, 20, 12,  4
 };


 __host__ __device__ void printSomething(int *d_PC1,int number){
     printf("printing something new \n");
     printf("%i ------ %i\n",d_PC1[10],5);
 }

 __global__ void doingSomething(int *d_PC1){
    printf("so then what\n");
    printf("%i\n",d_PC1[1]);
    printSomething(d_PC1,5);
 }

int main() {
    //printSomething();
    printf("hello world\n");
    printf("%i\n",PC1[0]);

    //host and device copies
    int *d_PC1;

    //size
    int sizePC1 = 56 * sizeof( int );

    //alloc space in cuda
    hipMalloc( (void **)&d_PC1, sizePC1 );

    //copy input to device
    hipMemcpy(d_PC1,PC1,sizePC1,hipMemcpyHostToDevice);

    //launch kernel on GPU
    doingSomething<<<1,1>>>(d_PC1);

    //free space
    hipFree(d_PC1);
    return 0;
}