
/**
 * calculate pi
 */

#include <stdio.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

//#define BLOCKS  512
#define NUMTHREADS 8192
#define ITERATIONS 2e09

/**
 * CUDA Kernel Device code
 * 
 */ 
/*****************************************************************************/

__global__ void calculatePi(double *piTotal, long int iterations, int totalThreads)
{   long int initIteration, endIteration;
    long int i = 0;
    double piPartial;
    
    int index = (blockDim.x * blockIdx.x) + threadIdx.x;

    initIteration = (iterations/totalThreads) * index;
    endIteration = initIteration + (iterations/totalThreads) - 1;
    
    i = initIteration;
    piPartial = 0;
    
    do{
        piPartial = piPartial + (double)(4.0 / ((i*2)+1));
        i++;
        piPartial = piPartial - (double)(4.0 / ((i*2)+1));
        i++;
    }while(i < endIteration);

    piTotal[index] = piPartial;
    
    __syncthreads();
    if(index == 0){
        for(i = 1; i < totalThreads; i++)
            piTotal[0] = piTotal[0] + piTotal[i];
    }
}


/******************************************************************************
 * Host main routine
 */
int main(int argc, char *argv[])
{   
    int blocksPerGrid = 20, threadsPerBlock, i, size;
    long int iterations;
    int totalThreads;
    double *h_pitotal, *d_pitotal;
    
    // sscanf(argv[1], "%i", &blocksPerGrid);
    hipError_t err = hipSuccess;

    size = sizeof(double)*NUMTHREADS;
    h_pitotal = (double *)malloc(size);
    if ( h_pitotal == NULL){
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    
    for(i = 0; i < NUMTHREADS; i++)
        h_pitotal[i] = 0.0;

    err = hipMalloc((void **)&d_pitotal, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_pitotal, h_pitotal, sizeof(double)*NUMTHREADS, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Lanzar KERNEL
    threadsPerBlock = NUMTHREADS/blocksPerGrid;
    totalThreads = blocksPerGrid * threadsPerBlock;
    iterations = ITERATIONS;
    printf("CUDA kernel launch with %d blocks of %d threads Total: %i\n", blocksPerGrid, threadsPerBlock, totalThreads  );
    calculatePi<<<blocksPerGrid, threadsPerBlock>>>(d_pitotal, iterations, totalThreads);
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_pitotal, d_pitotal, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_pitotal);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("\n%.12f", *h_pitotal);
    // Free host memory

    free(h_pitotal);
    err = hipDeviceReset();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return 0;
}

